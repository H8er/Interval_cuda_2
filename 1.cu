#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
using namespace std;
#include "cuda_interval_lib.h"
#include "cuda_check_error.cu"


#define TYPE float
typedef TYPE T;

__constant__ float dev_box[4];
// __constant__ int dev_threads[1];
// __constant__ int dev_blocks[1];
__constant__ int dev_n_of_ints[1];
__constant__ float dev_angle[1];

#include "functions.cu"



template<class T>
__global__ void large_grid(int* res){
  interval_gpu<T>* x = new interval_gpu<T>[dev_n_of_ints[0]];
  x[0] = interval_gpu<T>(dev_box[0] +  (threadIdx.x) * (dev_box[1] - dev_box[0])/blockDim.x,
                         dev_box[0] +(1+threadIdx.x) * (dev_box[1] - dev_box[0])/blockDim.x);
  x[1] = interval_gpu<T>(dev_box[2] +   (blockIdx.x) * (dev_box[3] - dev_box[2])/gridDim.x,
                         dev_box[2] + (1+blockIdx.x) * (dev_box[3] - dev_box[2])/gridDim.x);
  x[2] = interval_gpu<T>(M_PI/180*dev_angle[0], M_PI/180*(dev_angle[0]+1));
  res[(blockIdx.x*blockDim.x + threadIdx.x)] = 1;

  for(int i = 0; i < dev_n_of_func[0]; i++){
    res[(blockIdx.x*blockDim.x + threadIdx.x)] *= (*dev_func_pp[i])(x);
  }
}

void cout_objects(int* h_res, int blocks, int threads, float* host_box, double dx, double dy, float angle, int max_function_value){
for(int i = 0; i < blocks; i++){
  for(int j = 0; j < threads; j++){
    if(h_res[i * threads + j] % max_function_value > 0){
      interval_gpu<T> x3(host_box[0] + j*dx, host_box[0] + (j+1)*dx );
      interval_gpu<T> x4(host_box[2] + i*dy ,host_box[2] + (i+1)*dy );
      interval_gpu<T> xz(angle, angle + 1);
      cout<<x3<<":"<<x4<<":"<<xz<<"\n";
    }
  }
}
}

int main(int argc, char* argv[]){
cout<<fixed;
cout.precision(5);
int elapsed_seconds;

int n_of_ints = 3;
int n_of_func = 6;
int max_function_value = pow(2,n_of_func);
float host_box[4] = {-10,20,-10,20};
int blocks = 150;
double proportion = abs(host_box[1]-host_box[0])/abs(host_box[3]-host_box[2]);
int threads = blocks*proportion;
int second_grid = 100;
double dx = abs(host_box[1]-host_box[0])/threads;
double dy = abs(host_box[3]-host_box[2])/blocks;
double offset_x = host_box[0];
double offset_y = host_box[2];
int* res;
int* h_res   = (int*)malloc(sizeof(int)*blocks*threads);
int* h_res_2 = (int*)malloc(sizeof(int)*second_grid*second_grid);
int* corner  = (int*)malloc(sizeof(int));

float side_triangle = 2.0;
float xb1 = -side_triangle/2;
float yb1 =  side_triangle * sqrt(3)/6;
float xb2 =  side_triangle/2;
float yb2 =  side_triangle * sqrt(3)/6;
float xb3 =  0;
float yb3 =  side_triangle * sqrt(3)/3;
float host_triangle[6] = {xb1,yb1,xb2,yb2,xb3,yb3};
float host_angle;
int use_doublegrid = 1;

std::chrono::time_point<std::chrono:: high_resolution_clock> start, end;
start = std::chrono::high_resolution_clock::now();

for(int ang = -90; ang < 90; ang++){
    host_angle = ang;
    hipMalloc(&res, sizeof(int)*blocks*threads);
    hipMemcpyToSymbol(HIP_SYMBOL(dev_n_of_ints), &n_of_ints,     sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_n_of_func), &n_of_func,     sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_box),       &host_box,      sizeof(float)*4);
    hipMemcpyToSymbol(HIP_SYMBOL(triangle),      &host_triangle, sizeof(float)*6);
    hipMemcpyToSymbol(HIP_SYMBOL(dev_angle),     &host_angle,    sizeof(float));

    large_grid<T><<<blocks, threads>>>(res);
    CudaCheckError();
    hipDeviceSynchronize();
    hipMemcpy(h_res, res, sizeof(int)*blocks*threads, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    CudaCheckError();
    hipFree(res);
    hipDeviceReset();

    if(use_doublegrid == 0)
    {
      cout_objects(h_res, blocks, threads, host_box, dx,dy, host_angle, max_function_value);
    }
    else
    {
    for(int i = 0; i < blocks; i++){
      for(int j = 0; j < threads; j++){
        if(h_res[i * threads + j] % max_function_value > 0){
          host_box[0] = offset_x + j * dx;
          host_box[1] = offset_x + (j+1) * dx;
          host_box[2] = offset_y + i * dy;
          host_box[3] = offset_y + (i+1) * dy;
          double dx_1 = abs(host_box[1] - host_box[0]) / second_grid;
          double dy_1 = abs(host_box[3] - host_box[2]) / second_grid;

          hipDeviceReset();
          hipMalloc(&res, sizeof(int)*blocks*blocks);
          hipMemcpyToSymbol(HIP_SYMBOL(dev_n_of_ints), &n_of_ints,     sizeof(int));
          hipMemcpyToSymbol(HIP_SYMBOL(dev_n_of_func), &n_of_func,     sizeof(int));
          hipMemcpyToSymbol(HIP_SYMBOL(dev_box),       &host_box,      sizeof(float)*4);
          hipMemcpyToSymbol(HIP_SYMBOL(triangle),      &host_triangle, sizeof(float)*6);
          hipMemcpyToSymbol(HIP_SYMBOL(dev_angle),     &host_angle,    sizeof(float));
          large_grid<T><<<second_grid,second_grid>>>(res);

          CudaCheckError();
          hipDeviceSynchronize();
          hipMemcpy(h_res_2, res, sizeof(int)*second_grid*second_grid, hipMemcpyDeviceToHost);
          hipDeviceSynchronize();
          CudaCheckError();
          cout_objects(h_res_2, second_grid, second_grid, host_box, dx_1,dy_1, host_angle, max_function_value);
        }
    }
  }
}

hipDeviceReset();
}
    hipFree(res);
    hipFree(corner);
    hipFree(dev_n_of_ints);
    hipFree(dev_box);
    hipDeviceReset();

  end = std::chrono:: high_resolution_clock::now();
	elapsed_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
	std::time_t end_time = std::chrono::system_clock::to_time_t(end);
	cout<< "#. Время выполнения: " << elapsed_seconds << "  microseconds ~ "<<elapsed_seconds/1000000<<"sec\n";
  cout<< "#. Acc = "<<blocks << "x" <<threads<<"\n";

return 0;
}
