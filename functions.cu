#include "hip/hip_runtime.h"
// __device__ __host__ __forceinline__ int g1(interval_gpu<T> *x){
// interval_gpu<T> lmax(12);
// interval_gpu<T> f(x[0]*x[0] + x[1]*x[1] - lmax*lmax);
// return int(bool(f.upper() < 0) + bool(f.lower() < 0));
// }
// __device__ __host__ __forceinline__ int g2(interval_gpu<T> *x){
// interval_gpu<T> l(8);
// interval_gpu<T> f(l*l - x[0]*x[0] - x[1]*x[1]);
// return int(bool(f.upper() < 0) + bool(f.lower() < 0));
// }
//
// __device__ __host__ __forceinline__ int g3(interval_gpu<T> *x){
// interval_gpu<T> lmax(12);
// interval_gpu<T> l0(5);
// interval_gpu<T> f((x[0]-l0)*(x[0]-l0) + x[1]*x[1] - lmax*lmax);
// return int(bool(f.upper() < 0) + bool(f.lower() < 0));
// }
// __device__ __host__ __forceinline__ int g4(interval_gpu<T> *x){
// interval_gpu<T> l(8);
// interval_gpu<T> l0(5);
// interval_gpu<T> f(l*l  - (x[0]-l0)*(x[0]-l0) - x[1]*x[1]);
// return int(bool(f.upper() < 0) + bool(f.lower() < 0));
// }
//
// __constant__ int(*dev_func_pp[4])(interval_gpu<T>*) = {&g1,&g2,&g3,&g4};

#include <cmath>
__constant__ float triangle[6];

__device__ __forceinline__ int g1(interval_gpu<T> *x){
interval_gpu<T> lmax(6);
interval_gpu<T> xi(triangle[0]);
interval_gpu<T> yi(triangle[1]);
interval_gpu<T> xa1(0);
interval_gpu<T> ya1(0);
interval_gpu<T> rot_x(xi*cos(x[2]));
interval_gpu<T> rot_y(yi*sin(x[2]));
interval_gpu<T> f( square(x[0]+rot_x-rot_y-xa1) +
                   square(x[1]+rot_x+rot_y-ya1)-lmax*lmax );
return int(bool(f.upper() < 0) + bool(f.lower() < 0));
}
__device__ __forceinline__ int g2(interval_gpu<T> *x){
interval_gpu<T> lmin(1);
interval_gpu<T> xi(triangle[0]);
interval_gpu<T> yi(triangle[1]);
interval_gpu<T> xa1(0);
interval_gpu<T> ya1(0);
interval_gpu<T> rot_x(xi*cos(x[2]));
interval_gpu<T> rot_y(yi*sin(x[2]));
interval_gpu<T> f( lmin*lmin - square(x[0]+rot_x-rot_y-xa1) -
                   square(x[1]+rot_x+rot_y-ya1) );
return int(bool(f.upper() < 0) + bool(f.lower() < 0));
}

__device__ __forceinline__ int g3(interval_gpu<T> *x){
interval_gpu<T> lmax(6);
interval_gpu<T> xi(triangle[2]);
interval_gpu<T> yi(triangle[3]);
interval_gpu<T> xa2(6);
interval_gpu<T> ya2(0);
interval_gpu<T> rot_x(xi*cos(x[2]));
interval_gpu<T> rot_y(yi*sin(x[2]));
interval_gpu<T> f( square(x[0]+rot_x-rot_y-xa2) +
                   square(x[1]+rot_x+rot_y-ya2)-lmax*lmax );
return int(bool(f.upper() < 0) + bool(f.lower() < 0));
}
__device__ __forceinline__ int g4(interval_gpu<T> *x){
interval_gpu<T> lmin(1);
interval_gpu<T> xi(triangle[2]);
interval_gpu<T> yi(triangle[3]);
interval_gpu<T> xa2(6);
interval_gpu<T> ya2(0);
interval_gpu<T> rot_x(xi*cos(x[2]));
interval_gpu<T> rot_y(yi*sin(x[2]));
interval_gpu<T> f( lmin*lmin - square(x[0]+rot_x-rot_y-xa2) -
                   square(x[1]+rot_x+rot_y-ya2) );
return int(bool(f.upper() < 0) + bool(f.lower() < 0));
}

__device__ __forceinline__ int g5(interval_gpu<T> *x){
interval_gpu<T> lmax(6);
interval_gpu<T> xi(triangle[4]);
interval_gpu<T> yi(triangle[5]);
interval_gpu<T> xa3(3);
interval_gpu<T> ya3(6*1.7320508076/2);
interval_gpu<T> rot_x(xi*cos(x[2]));
interval_gpu<T> rot_y(yi*sin(x[2]));
interval_gpu<T> f( square(x[0]+rot_x-rot_y-xa3) +
                   square(x[1]+rot_x+rot_y-ya3)-lmax*lmax );
return int(bool(f.upper() < 0) + bool(f.lower() < 0));
}
__device__ __forceinline__ int g6(interval_gpu<T> *x){
interval_gpu<T> lmin(1);
interval_gpu<T> xi(triangle[4]);
interval_gpu<T> yi(triangle[5]);
interval_gpu<T> xa3(3);
interval_gpu<T> ya3(6*1.7320508076/2);
interval_gpu<T> rot_x(xi*cos(x[2]));
interval_gpu<T> rot_y(yi*sin(x[2]));
interval_gpu<T> f( lmin*lmin - square(x[0]+rot_x-rot_y-xa3) -
                   square(x[1]+rot_x+rot_y-ya3) );
return int(bool(f.upper() < 0) + bool(f.lower() < 0));
}

__constant__ int dev_n_of_func[1];
__constant__ int(*dev_func_pp[6])(interval_gpu<T>*) = {
  &g1,&g2,
  &g3,&g4,
  &g5,&g6
};
